#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <string.h> 
#include <stddef.h> 
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


#define N 100
#define MAX_LEVELS 10
#define TOLERANCE 0.0001

__global__ void init_pade(double *A){
    int i = blockIdx.x, j = threadIdx.x;
    if (i == j){
        if (i==0 || i==(N-1))
            A[i*N + j] = 1.0;

        else 
            A[i*N + j] = 4.0;
    }
    else if (abs(i - j) == 1){
        if (i==0 || i==(N-1))
            A[i*N + j] = 2.0;

        else 
            A[i*N + j] = 1.0;
    }
    else
        A[i*N + j] = 0.0;
}

__global__ void init_tridiag(double *A){
    int i = blockIdx.x, j = threadIdx.x;
    if (i == j)
        A[i*N + j] = 2.0;
    else if (abs(i - j) == 1)
        A[i*N + j] = -1.0;
    else
        A[i*N + j] = 0.0;
}

__global__ void init_b(double *b, double *f){
    double delta = (double)3/N;
    for(int i=0;i<N;i++)
    {
        f[i]=sin(5*i*delta);
        printf("%f , ",f[i]);
    }

    for (int i = 1; i < N-1; i++) {
        b[i] = 3*(f[i+1]-f[i-1])/delta;
    }

    b[N-1]=(2.5*f[N-1]-2*f[N-2]-0.5*f[N-3])/delta;
    b[0]=(-2.5*f[0]+2*f[1]+0.5*f[2])/delta;
}

__global__ void init_b(double* b){
    int i = threadIdx.x;
    b[i] = 1;
}

__global__ void init_M(double* M, double *restriction, double *A){
    int i = blockIdx.x, j = threadIdx.x;
    for (int k=0; k<N; k++) {
        M[i*N + j] += restriction[i*N + k]*A[k*N + j];
    }
}

__global__ void init_pro_rest(double *prolongation, double *restriction){
    int i = blockIdx.x, j = threadIdx.x;
    if ((i%2)!=0 && j==(i-1)/2)
        prolongation[i*(N/2) + j]=1;
    if ((i%2)==0 && j==(i/2))
        prolongation[i*(N/2) + j]=1;
    else 
        prolongation[i*(N/2) + j]=0;

    restriction[j*N + i]=prolongation[i*(N/2) + j];
}

__global__ void init_coarse(double *A, double *A_coarse){
    int i = blockIdx.x, j = threadIdx.x;
    A_coarse[(i/2)*(N/2)+(j/2)]= A[i*N+j]+ A[(i+1)*N+j]+A[i*N+j+1]+A[(i+1)*N+j+1];
}

__global__ void init_coarse(double *A_coarse, double *M, double *prolongation){
    int i = blockIdx.x, j = threadIdx.x;
    for( int k=0; k<N; k++ ){
        A_coarse[i*(N/2)+j]+= M[i*N+k]*prolongation[k*(N/2) + j];
    }
}

__global__ void gauss_seidel(double *A, double *b, double *x, double* residual, bool fl) {
    int i = threadIdx.x;
    double sum = b[i];
    for ( j = 0; j < N; j++) {
        if (j != i)
            sum -= A[i*N+j] * x[j];
    }
    if(fl)
        x[i] = sum / A[i*N+i];
    else
        residual[i] = sum;
}


__global__ void gauss_seidel_coarse(double *A_coarse, double *residual_coarse, double *correction_coarse) {
    int i = threadIdx.x;
    double sum = residual_coarse[i];
    for ( j = 0; j < N/2; j++) {
        if (j != i)
            sum -= A_coarse[i*(N/2)+j] * correction_coarse [j];
    }
    correction_coarse [i] = sum / A_coarse[i*(N/2)+i];
}

__global__ void calculate_res(double* residual_coarse, double* restriction, double* residual){
    int i = blockIdx.x, k = threadIdx.x;
    residual_coarse[i] += restriction[i*N+k]*residual[k];
}

__global__ void calculate_cor(double* correction, double* prolongation, double* correction_coarse){
    int i = blockIdx.x, k = threadIdx.x;
    correction[i]= prolongation[i*(N/2)+k]*correction_coarse[k];
}

__global__ void update_x(double* x, double* prev_x, double* correction, bool fl){
    int i = threadIdx.x;
    if(fl)
        x[i] += correction[i];
    else
        prev_x[i] = x[i];
}

int main(int argc, char* argv[]) {
    hipMalloc((void**)&A, N*N*sizeof(double));
    hipMalloc((void**)&b, N*sizeof(double));
    hipMalloc((void**)&x, N*sizeof(double));
    hipMalloc((void**)&restriction, (N/2)*N*sizeof(double));
    hipMalloc((void**)&prolongation, N*(N/2)*sizeof(double));
    hipMalloc((void**)&A_coarse, (N/2)*(N/2)*sizeof(double));
    hipMemset(x, 0, N*sizeof(double));

    init_pade<<< N,N >>>(A);
    hipMalloc((void**)&f, N*sizeof(double));
    init_b<<< 1,1 >>>(b,f);

    // init_tridiag<<< N,N >>>(A);
    // init_b<<< 1,N >>>(b);

    init_pro_rest<<< N,N/2 >>>(prolongation, restriction);

    // hipMalloc((void**)&M, (N/2)*N*sizeof(double));
    // init_M<<< N/2,N >>>(M,restriction,A);
    // init_coarse<<< N/2,N/2 >>>(A_coarse,M,prolongation)

    init_coarse<<< N,N >>>(A, A_coarse);

    hipMalloc((void**)&residual, N*sizeof(double));
    hipMalloc((void**)&correction, N*sizeof(double));
    hipMalloc((void**)&residual_coarse, (N/2)*sizeof(double));
    hipMalloc((void**)&correction_coarse, (N/2)*sizeof(double));
    hipMalloc((void**)&prev_x, N*sizeof(double));
    hipMalloc((void**)&error, sizeof(double));
 
    gauss_seidel<<< 1,N >>>(A, b, x, residual, true);

    do {
        update_x<<< 1,N >>>(x, correction, false);
        gauss_seidel<<< 1,N >>>(A, b, x, residual, false);
       
        for (int level = 1; level < MAX_LEVELS; level++) {
            hipMemset(residual_coarse, 0, (N/2)*sizeof(double));
            hipMemset(correction_coarse, 0, (N/2)*sizeof(double));
            hipMemset(correction, 0, N*sizeof(double));
            calculate_res<<< N/2,N >>>(residual_coarse,restriction,residual);
            
            gauss_seidel_coarse<<< 1,N/2 >>>(A_coarse, residual_coarse, correction_coarse);

            calculate_cor<<< N,N/2 >>>(correction,prolongation,correction_coarse);
            update_x<<< 1,N >>>(x, correction, true);

            gauss_seidel<<< 1,N >>>(A, b, x, residual, true);
            gauss_seidel<<< 1,N >>>(A, b, x, residual, false);
        }

        hipMemset(error, 0, sizeof(double));
        for (int i = 0; i < N; i++) {
            error += fabs(x[i] - prev_x[i]);
        }
        // printf("Final x vector:\n");
        // for (int i = 0; i < N; i++) {
        //     printf("%.6f , ", x[i]);
        // }printf("\n");

    } while (error > TOLERANCE);

    double x_host[N];
    hipMemcpy(x_host, x, N*sizeof(double), hipMemcpyDeviceToHost);
    printf("Final x vector:\n");
    for (int i = 0; i < N; i++) {
        printf("%.6f , ", x_host[i]);
    }

  return 0;
}